#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "cudnn/common/common.cuh"
#include "cudnn/common/cuda_common.hpp"
#include "cudnn/common/slice_impl.cuh"

namespace Cudnn {

namespace {
constexpr int kNumElementsPerThread = GridDim::maxElementsPerThread;
constexpr int kNumThreadsPerBlock = GridDim::maxThreadsPerBlock;
}  // namespace

template <bool is_grad, int DIMS, typename T>
__global__ void _SliceKernel(const TArray<int64_t> starts,
                             const TArray<int64_t> steps,
                             const TArray<int64_t> input_strides,
                             const TArray<fast_divmod> output_strides,
                             const T* input_data, T* output_data,
                             const CUDA_LONG N) {
  CUDA_LONG start =
      kNumElementsPerThread * kNumThreadsPerBlock * blockIdx.x + threadIdx.x;
  T values[kNumElementsPerThread];
  CUDA_LONG id;
  if (is_grad) {
    id = start;
#pragma unroll
    for (int i = 0; i < kNumElementsPerThread; ++i) {
      if (id < N) {
        values[i] = input_data[id];
        id += kNumThreadsPerBlock;
      }
    }
  }

  id = start;
#pragma unroll
  for (int i = 0; i < kNumElementsPerThread; ++i) {
    if (id < N) {
      CUDA_LONG input_index = 0;
      int div;
      int mod = id;
      int dim = 0;
#pragma unroll
      for (; dim < DIMS - 1; ++dim) {
        output_strides[dim].divmod(mod, div, mod);
        input_index += (starts[dim] + div * steps[dim]) * input_strides[dim];
      }
      input_index += starts[dim] + mod * steps[dim];
      if (is_grad) {
        output_data[input_index] = values[i];
      } else {
        values[i] = input_data[input_index];
      }
      id += kNumThreadsPerBlock;
    }
  }

  if (!is_grad) {
    id = start;
#pragma unroll
    for (int i = 0; i < kNumElementsPerThread; ++i) {
      if (id < N) {
        output_data[id] = values[i];
        id += kNumThreadsPerBlock;
      }
    }
  }
}

template <bool is_grad>
bool SliceImplEx(hipStream_t stream, const size_t element_size,
                 const int32_t dimension_count, const TArray<int64_t>& starts,
                 const TArray<int64_t>& steps,
                 const TArray<int64_t>& input_strides,
                 const TArray<fast_divmod>& output_strides,
                 const void* input_data, void* output_data, const size_t N) {
  int blocksPerGrid =
      static_cast<int>(CeilDiv(N, kNumThreadsPerBlock * kNumElementsPerThread));
  switch (element_size) {
#define HANDLE_DIMS(ELEMENT_TYPE, DIMS)                                    \
  case DIMS: {                                                             \
    _SliceKernel<is_grad, DIMS, ELEMENT_TYPE>                              \
        <<<blocksPerGrid, kNumThreadsPerBlock, 0, stream>>>(               \
            starts, steps, input_strides, output_strides,                  \
            reinterpret_cast<const ToCudaType<ELEMENT_TYPE>::MappedType*>( \
                input_data),                                               \
            reinterpret_cast<ToCudaType<ELEMENT_TYPE>::MappedType*>(       \
                output_data),                                              \
            (CUDA_LONG)N);                                                 \
  } break
#define HANDLE_ELEMENT_TYPE(ELEMENT_TYPE) \
  case sizeof(ELEMENT_TYPE): {            \
    switch (dimension_count) {            \
      HANDLE_DIMS(ELEMENT_TYPE, 1);       \
      HANDLE_DIMS(ELEMENT_TYPE, 2);       \
      HANDLE_DIMS(ELEMENT_TYPE, 3);       \
      HANDLE_DIMS(ELEMENT_TYPE, 4);       \
      HANDLE_DIMS(ELEMENT_TYPE, 5);       \
      HANDLE_DIMS(ELEMENT_TYPE, 6);       \
      HANDLE_DIMS(ELEMENT_TYPE, 7);       \
      HANDLE_DIMS(ELEMENT_TYPE, 8);       \
    }                                     \
  } break
    HANDLE_ELEMENT_TYPE(int8_t);
    HANDLE_ELEMENT_TYPE(int16_t);
    HANDLE_ELEMENT_TYPE(int32_t);
    HANDLE_ELEMENT_TYPE(int64_t);
    default:
      return false;
#undef HANDLE_ELEMENT_TYPE
#undef HANDLE_DIMS
  }

  return true;
}

bool SliceImpl(hipStream_t stream, const size_t element_size,
               const int32_t dimension_count, const TArray<int64_t>& starts,
               const TArray<int64_t>& steps,
               const TArray<int64_t>& input_strides,
               const TArray<fast_divmod>& output_strides,
               const void* input_data, void* output_data, const size_t N) {
  return SliceImplEx<false>(stream, element_size, dimension_count, starts,
                            steps, input_strides, output_strides, input_data,
                            output_data, N);
}

}  // namespace Cudnn
